#include "hip/hip_runtime.h"
﻿#include"hip/hip_runtime.h"
#include"hip/device_functions.h"
#include""
#include<stdio.h>
#include<iostream>

#define N 2000
#define BLOCK_SIZE 1024
float elm[N][N] = { 0.0 };

using namespace std;

__global__ void division_kernel(float* data, int k) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;//计算线程索引
	while (tid < N) {
		int element = data[k * N + k];
		int temp = data[k * N + tid];
		//请同学们思考，如果分配的总线程数小于 N 应该怎么办？
		data[k * N + tid] = (float)temp / element;
		r++;
		tid += r * blockDim.x;//计算线程索引
	}

	return;
}

__global__ void eliminate_kernel(float* data, int k) {
	int tx = blockDim.x * blockIdx.x + threadIdx.x;
	if (tx == 0)
		data[k * N + k] = 1.0;//对角线元素设为 1
	int row = k + 1 + blockIdx.x;//每个块负责一行
	while (row < N) {
		int tid = threadIdx.x;
		while (k + 1 + tid < N) {
			int col = k + 1 + tid;
			float temp_1 = data[(row * N) + col];
			float temp_2 = data[(row * N) + k];
			float temp_3 = data[k * N + col];
			data[(row * N) + col] = temp_1 - temp_2 * temp_3;
			tid = tid + blockDim.x;
		}
		__syncthreads();//块内同步
		if (threadIdx.x == 0) {
			data[row * N + k] = 0;
		}
		row += gridDim.x;
	}
	return;
}

void display() {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			cout << elm[i][j] << " ";
		}
		cout << endl;
	}
}
void init() {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			elm[i][j] = 0.0;
		}
	}
	for (int i = 0; i < N; i++) {
		elm[i][i] = 1.0;
	}
	for (int r = 0; r < 5 * N; r++) {
		int i1 = rand() % N;
		int i2 = rand() % N;
		float rate = rand() % 10 / 10.0;;
		if (i1 != i2) {
			for (int j = 0; j < N; j++) {
				elm[i1][j] += rate * elm[i2][j];
			}
		}
	}
	//display();
}




extern "C" float* paraCuda() {
	init();
	float* temp = new float[N * N];
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			temp[i * N + j] = elm[i][j];
		}
	}
	hipError_t ret;//用于错误检查，当 CUDA 接口调用成功会返回 cudaSucess
	float* gpudata;
	float* result = new float[N * N];
	int size = N * N * sizeof(float);

	ret = hipMalloc(&gpudata, size);//分配显存空间
	if (ret != hipSuccess) {
		printf("hipMalloc gpudata failed!\n");
	}
	ret = hipMemcpy(gpudata, temp, size, hipMemcpyHostToDevice);//将数据传输至 GPU 端
	if (ret != hipSuccess) {
		printf("hipMemcpyHostToDevice failed!\n");
	}
	dim3 dimBlock(BLOCK_SIZE, 1);//线程块
	dim3 dimGrid(1, 1);//线程网格
	hipEvent_t start, stop;//计时器
	float elapsedTime = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);//开始计时

	for (int k = 0; k < N; k++) {
		division_kernel << <dimGrid, dimBlock >> > (gpudata, k);//负责除法任务的核函数
		hipDeviceSynchronize();//CPU 与 GPU 之间的同步函数
		ret = hipGetLastError();
		if (ret != hipSuccess) {
			printf("division_kernel failed, %s\n", hipGetErrorString(ret));
		}
		eliminate_kernel << <dimGrid, dimBlock >> > (gpudata, k);//负责消去任务的核函数
		hipDeviceSynchronize();
		ret = hipGetLastError();
		if (ret != hipSuccess) {
			printf("eliminate_kernel failed, %s\n", hipGetErrorString(ret));
		}
	}


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);//停止计时
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("GPU_LU:%f ms\n", elapsedTime);
	hipError_t cudaStatus2 = hipGetLastError();
	if (cudaStatus2 != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus2));
	}
	ret = hipMemcpy(result, gpudata, size, hipMemcpyDeviceToHost);//将数据传回 CPU 端
	if (ret != hipSuccess) {
		printf("hipMemcpyDeviceToHost failed!\n");
	}
	hipFree(gpudata);//释放显存空间，用 CUDA 接口分配的空间必须用 hipFree 释放
	//销毁计时器
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return result;
}